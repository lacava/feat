
#include <hip/hip_runtime.h>
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
static void HandleError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

